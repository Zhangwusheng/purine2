#include "hip/hip_runtime.h"
// Copyright Lin Min 2015
#include "operations/include/drop.hpp"
#include "caffeine/caffeine.hpp"
#include <hip/hip_runtime.h>

namespace purine {

    Drop::Drop(const vector<Tensor*>& inputs,
            const vector<Tensor*>& outputs, const param_tuple& args)
        : Operation(inputs, outputs) {
            std::tie(rate_, dropVector_, isTest_) = args;
            dDropVector_ = NULL;
            //CHECK_EQ(inputs_[0]->size(), outputs_[0]->size());
        }

    Drop::~Drop() {
    }

    void Drop::compute_cpu(const vector<bool>& add){
        printf("Drop::Compute_cpu\n");
    }

    struct cuSize{
        int n, c, h, w;
    };

    struct cuOffset{
        int n, c, h, w;
    };

    struct cuStride{
        int n,c,h,w;
    };

    cuSize getCuSize( Size size ){
        cuSize ret;
        ret.n = size.num();
        ret.c = size.channels();
        ret.w = size.width();
        ret.h = size.height();
        return ret;
    }

    cuOffset getCuOffset( Offset offset ){
        cuOffset ret;
        ret.n = offset.noffset();
        ret.c = offset.coffset();
        ret.h = offset.hoffset();
        ret.w = offset.woffset();
        return ret;
    }

    cuStride getCuStride( Stride stride ){
        cuStride ret;
        ret.n = stride.nstride();
        ret.c = stride.cstride();
        ret.h = stride.hstride();
        ret.w = stride.wstride();
        return ret;
    }

    __device__ inline int getNStride(cuSize size){
        return size.c * size.h * size.w;
    }

    __device__ inline int getCStride(cuSize size){
        return size.h * size.w;
    }

    __device__ inline int getHStride(cuSize size){
        return size.w;
    }

    __device__ inline int getN(int index, cuSize size){
        int cur = index / getNStride(size);
        return cur;
    }

    __device__ inline int getC(int index, cuSize size){
        int cur = index % getNStride(size);
        cur = cur / getCStride(size);
        return cur;
    }

    __device__ inline int getH(int index, cuSize size){
        int cur = index % getCStride(size);
        cur = cur / getHStride(size);
        return cur;
    }

    __device__ inline int getW(int index, cuSize size){
        int cur = index % size.w;
        return cur;
    }

    __device__ inline int getIndex(int n, int c, int h, int w, cuOffset offset, cuStride stride){
        int n1 = n + offset.n;
        int c1 = c + offset.c;
        int h1 = h + offset.h;
        int w1 = w + offset.w;
        return n1 * stride.n + c1 * stride.c + h1 * stride.h + w1 * stride.w;
    }
    /*
       drop forward
     */
    __global__ void DropForward(const DTYPE* in, cuSize input_size, cuOffset input_offset, cuStride input_stride,
            DTYPE* out, cuSize output_size, cuOffset output_offset, cuStride output_stride,
            float* dropVector, 
            int data_size){
        int thread_index = threadIdx.x + blockIdx.x * blockDim.x;
        int num_threads = blockDim.x * gridDim.x;
        for(int i = 0; i < data_size; i += num_threads){
            int index = i + thread_index;

            if(index < data_size){
                int n = getN(index, input_size);
                int c = getC(index, input_size);
                int h = getH(index, input_size);
                int w = getW(index, input_size);
                int input_index = getIndex(n,c,h,w, input_offset, input_stride);
                int output_index = getIndex(n,c,h,w, output_offset, output_stride);
                /*
                if(input_index != output_index){
                    printf("haahah\n");
                }
                if(input_index != index){
                    printf("hihihi\n");
                }
                if( n * input_size.c + c > input_size.n * input_size.c) {
                    printf("dropvector error\n");
                }
                */
                
                //out[index] = in[index] * alpha;
                //out[output_index] = in[input_index] * alpha;
                /*
                if( dropVector[n * input_size.c + c] < 0.001){
                    printf("papapu\n");
                }*/
                out[output_index] = in[input_index] * dropVector[n * input_size.c + c];
                //out[index] = in[index];

                //out[output_index] = in[input_index];
                /*if(input_index != index){
                    printf("forword wocaocaocao\n");
                }*/
                /*if(index > data_size - 50){
                  printf("%d %d %d %d\n", n, c, h, w);
                  }*/
                //if(index < 50){
                //    printf("%f %f alpha%f\n", out[index], in[index], alpha);
                //}
            }
        }
    }

    void printStride(cuStride stride){
        printf("stride %d %d %d %d\n", stride.n, stride.c, stride.h, stride.w);
    }

    void printOffset(cuOffset offset){
        printf("offset%d %d %d %d\n", offset.n, offset.c, offset.h, offset.w);
    }

    void printSize( cuSize size){
        printf("size %d %d %d %d\n", size.n, size.c, size.h, size.w);
    }

    void Drop::compute_gpu(const vector<bool>& add) {
        std::lock_guard<std::mutex>lock_guard_(outputs_[0]->get_mutex());
        Size s = inputs_[0]->size();
        //CHECK_EQ(inputs_[0]->size().count(), outputs_[0]->size().count());
        //CHECK_EQ(inputs_[1]->size().count(), inputs_[0]->size().count());
        int data_size = s.count();
        int num_blocks  = CAFFE_GET_BLOCKS(data_size);
        int num_threads = CAFFE_CUDA_NUM_THREADS;

        int nSizeOfFeature = s.num() * s.channels();
        if( isTest_ == true ){
            for( int i = 0; i < nSizeOfFeature; i++){
                dropVector_[i] = 1.0 - rate_; 
            }
        }
        else{
            for(int i = 0; i < nSizeOfFeature; i++){
                float tmp = 1.0f * rand() / RAND_MAX;
                if( rate_ <= tmp ){
                    dropVector_[i] = 1.0;
                }
                else{
                    dropVector_[i] = 0.0;
                }
            }
        }

        if(dDropVector_ == NULL)
            hipMalloc((void**)&dDropVector_, nSizeOfFeature * sizeof(float));
        hipMemcpy(dDropVector_, dropVector_, nSizeOfFeature * sizeof(float), hipMemcpyHostToDevice);  

        //printf("up alpha %d %f rate %f \n", rate_, alpha, rate_[0]);
        cuSize input_size = getCuSize(inputs_[0]->size());
        cuOffset input_offset = getCuOffset(inputs_[0]->offset());
        cuStride input_stride = getCuStride(inputs_[0]->stride());

        cuSize output_size = getCuSize(outputs_[0]->size());
        cuOffset output_offset = getCuOffset(outputs_[0]->offset());
        cuStride output_stride = getCuStride(outputs_[0]->stride());
        /*
           printSize(input_size);
           printSize(output_size);
           printOffset(input_offset);
           printOffset(output_offset);
           printStride(input_stride);
           printStride(output_stride);
           printf("\n\n");
         */
        //hipMemcpy(outputs_[0]->mutable_gpu_data(), inputs_[0]->gpu_data(), data_size * sizeof(float), hipMemcpyDeviceToDevice);
        DropForward<<<num_blocks, num_threads, 0, stream()>>> 
            (inputs_[0]->gpu_data(), input_size, input_offset, input_stride,
             outputs_[0]->mutable_gpu_data(), output_size, output_offset, output_stride,
             dDropVector_, 
             data_size); 
        CUDA_POST_KERNEL_CHECK;
        (hipDeviceSynchronize());
    }

    DropDown::DropDown(const vector<Tensor*>& inputs,
            const vector<Tensor*>& outputs, const param_tuple& args)
        : Operation(inputs, outputs) {
            std::tie(rate_, dropVector_) = args;
            dDropVector_ = NULL;
            //CHECK_EQ(inputs_[0]->size(), outputs_[0]->size());
        }

    DropDown::~DropDown() {
    }

    void DropDown::compute_cpu(const vector<bool>& add){
        printf("wocao\n");
    }
    /*
       drop backward
       B{ top_[1], top_[0], bottom_[0] } >> *activation_down >> B{ bottom_[1] }; 
     */
    __global__ void DropBackward(
            const DTYPE* in_diff, cuSize in_diff_size, cuOffset in_diff_offset, cuStride in_diff_stride,
            const DTYPE* in_data, cuSize in_data_size, cuOffset in_data_offset, cuStride in_data_stride,
            DTYPE* out_diff, cuSize out_diff_size, cuOffset out_diff_offset, cuStride out_diff_stride, 
            float* dropVector,
            int data_size){

        int thread_index = threadIdx.x + blockIdx.x * blockDim.x;
        int num_threads = blockDim.x * gridDim.x;

        for(int i = 0; i < data_size; i += num_threads){
            int index = i + thread_index;
            if(index < data_size){
                int n = getN(index, in_diff_size);
                int c = getC(index, in_diff_size);
                int h = getH(index, in_diff_size);
                int w = getW(index, in_diff_size);
                int in_diff_index = getIndex(n,c,h,w, in_diff_offset, in_diff_stride);
                int in_data_index = getIndex(n,c,h,w, in_data_offset, in_data_stride);
                int output_diff_index = getIndex(n,c,h,w, out_diff_offset, out_diff_stride);
                //out_diff[output_diff_index] = in_diff[in_diff_index] * alpha;
                //out_diff[index] = in_diff[index] * alpha;
                /*
                if( output_diff_index != in_diff_index ){
                    printf("backward  output != index");
                }
                */
                //out_diff[output_diff_index] = in_diff[in_diff_index] * alpha;
                out_diff[output_diff_index] = in_diff[in_diff_index] * dropVector[n * in_diff_size.c + c];
                /*
                   if( dropVector[n * in_diff_size.c + c] < 0.001){
                   printf("papapu\n");
                   }*/

                /*
                   if( output_diff_index != index){
                   printf("co\n");
                   }
                   if( output_diff_index != in_diff_index){
                   printf("ho\n");
                   }*/

                /*if(index < 50){
                //printf("%f %f %f\n", out_diff[output_diff_index], in_diff[in_diff_index], in_data[in_data_index]);
                }*/
            }
        }
    }

    /*
       lrelu backward
       B{ top_[1], top_[0], bottom_[0] } >> *activation_down >> B{ bottom_[1] }; 
     */
    void DropDown::compute_gpu(const vector<bool>& add){
        std::lock_guard<std::mutex>lock_guard_(outputs_[0]->get_mutex());
        CHECK_EQ(inputs_[0]->size().count(), outputs_[0]->size().count());
        Size s = inputs_[0]->size();
        int data_size = s.count();
        int num_blocks  = CAFFE_GET_BLOCKS(data_size);
        int num_threads = CAFFE_CUDA_NUM_THREADS;

        cuSize input0_size = getCuSize(inputs_[0]->size());
        cuOffset input0_offset = getCuOffset(inputs_[0]->offset());
        cuStride input0_stride = getCuStride(inputs_[0]->stride());

        cuSize input1_size = getCuSize(inputs_[1]->size());
        cuOffset input1_offset = getCuOffset(inputs_[1]->offset());
        cuStride input1_stride = getCuStride(inputs_[1]->stride());

        cuSize output_size = getCuSize(outputs_[0]->size());
        cuOffset output_offset = getCuOffset(outputs_[0]->offset());
        cuStride output_stride = getCuStride(outputs_[0]->stride());


        int nSizeOfFeature = s.num() * s.channels();
        if(dDropVector_ == NULL)
            hipMalloc((void**)&dDropVector_, nSizeOfFeature * sizeof(float));
        hipMemcpy(dDropVector_, dropVector_, nSizeOfFeature * sizeof(float), hipMemcpyHostToDevice);  

        /*printSize(input0_size);
          printSize(input1_size);
          printSize(output_size);

          printOffset(input0_offset);
          printOffset(input1_offset);
          printOffset(output_offset);

          printStride(input0_stride);
          printStride(input1_stride);
          printStride(output_stride);
          printf("\n\n");
         */
        // hipMemcpy(outputs_[0]->mutable_gpu_data(), inputs_[0]->gpu_data(), data_size * sizeof(float), hipMemcpyDeviceToDevice);
        DropBackward<<< num_blocks, num_threads, 0, stream()>>>(
                inputs_[0]->gpu_data(), input0_size, input0_offset, input0_stride,
                inputs_[1]->gpu_data(), input1_size, input1_offset, input1_stride,
                outputs_[0]->mutable_gpu_data(), output_size, output_offset, output_stride,
                dDropVector_, 
                data_size);
        CUDA_POST_KERNEL_CHECK;
        (hipDeviceSynchronize());
    }

}
