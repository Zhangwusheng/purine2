#include "hip/hip_runtime.h"
// Copyright Lin Min 2015
#include "operations/include/activation.hpp"
#include "caffeine/caffeine.hpp"

namespace purine {

    Activation::Activation(const vector<Tensor*>& inputs,
            const vector<Tensor*>& outputs, const param_tuple& args)
        : Operation(inputs, outputs) {
            std::tie(mode_) = args;
            CHECK_EQ(inputs_[0]->size(), outputs_[0]->size());
            Size bottom_size = inputs_[0]->size();
            Stride bottom_stride = inputs_[0]->stride();
            Size top_size = outputs_[0]->size();
            Stride top_stride = outputs_[0]->stride();
            cudnn::createTensor4dDesc<DTYPE>(&bottom_desc_, bottom_size, bottom_stride);
            cudnn::createTensor4dDesc<DTYPE>(&top_desc_, top_size, top_stride);
            if (mode_ == "relu") {
                activation_mode_ = HIPDNN_ACTIVATION_RELU;
            } else if (mode_ == "sigmoid") {
                activation_mode_ = HIPDNN_ACTIVATION_SIGMOID;
            } else if (mode_ == "tanh") {
                activation_mode_ = HIPDNN_ACTIVATION_TANH;
            } else if (mode_ == "lrelu"){
            } else{
                LOG(FATAL) << "Unknown activation mode " << mode_;
            }
        }

    Activation::~Activation() {
        CUDNN_CHECK(hipdnnDestroyTensorDescriptor(bottom_desc_));
        CUDNN_CHECK(hipdnnDestroyTensorDescriptor(top_desc_));
    }

    /*
       lrelu forward
     */
    __global__ void LreluForward(const DTYPE* in, DTYPE* out, int data_size){
        int thread_index = threadIdx.x + blockIdx.x * blockDim.x;
        int num_threads = blockDim.x * gridDim.x;
        for(int i = 0; i < data_size; i += num_threads){
            int index = i + thread_index;
            out[index] = in[index] > 0 ? in[index] : in[index] * 0.01;
        }
    }

    void Activation::compute_gpu(const vector<bool>& add) {
        
        if(mode_ == "lrelu"){
            Size s = inputs_[0]->size();
            int data_size = s.num() * s.channels() * s.height() * s.width();
            int num_blocks  = std::min(CAFFE_GET_BLOCKS(data_size), 1024);
            int num_threads = CAFFE_CUDA_NUM_THREADS;
            LreluForward<<<num_blocks, num_threads, 0, stream()>>> 
                (inputs_[0]->gpu_data(), 
                    outputs_[0]->mutable_gpu_data(),
                    data_size); 
            CUDA_POST_KERNEL_CHECK;
        }
        else if(mode_ == "relu" || mode_ == "sigmoid" || mode_ == "tanh"){
            DTYPE alpha = 1.;
            DTYPE beta = add[0] ? 1. : 0.;
            CUDNN_CHECK(hipdnnActivationForward(cudnn_handle(), activation_mode_,
                        &alpha, bottom_desc_, inputs_[0]->gpu_data(), &beta, top_desc_,
                        outputs_[0]->mutable_gpu_data()));
        }
        else {
            LOG(FATAL) << "Unknown activation mode " << mode_;
        }
    }

    ActivationDown::ActivationDown(const vector<Tensor*>& inputs,
            const vector<Tensor*>& outputs, const param_tuple& args)
        : Operation(inputs, outputs) {
            std::tie(mode_) = args;
            CHECK_EQ(inputs_[0]->size(), outputs_[0]->size());
            Size bottom_size = outputs_[0]->size();
            Stride bottom_stride = outputs_[0]->stride();
            Size top_size = inputs_[0]->size();
            Stride top_stride = inputs_[0]->stride();
            cudnn::createTensor4dDesc<DTYPE>(&bottom_desc_, bottom_size, bottom_stride);
            cudnn::createTensor4dDesc<DTYPE>(&top_desc_, top_size, top_stride);
            if (mode_ == "relu") {
                activation_mode_ = HIPDNN_ACTIVATION_RELU;
            } else if (mode_ == "sigmoid") {
                activation_mode_ = HIPDNN_ACTIVATION_SIGMOID;
            } else if (mode_ == "tanh") {
                activation_mode_ = HIPDNN_ACTIVATION_TANH;
            } else if (mode_ == "lrelu"){
            }
            else {
                LOG(FATAL) << "Unknown activation mode " << mode_;
            }
        }

    ActivationDown::~ActivationDown() {
        CUDNN_CHECK(hipdnnDestroyTensorDescriptor(bottom_desc_));
        CUDNN_CHECK(hipdnnDestroyTensorDescriptor(top_desc_));
    }

    /*
       lrelu backward
       B{ top_[1], top_[0], bottom_[0] } >> *activation_down >> B{ bottom_[1] }; 
     */
    __global__ void LreluBackward(const DTYPE* in_diff, const DTYPE* in_data, 
            DTYPE* out_diff, int data_size){
        
        int thread_index = threadIdx.x + blockIdx.x * blockDim.x;
        int num_threads = blockDim.x * gridDim.x;

        for(int i = 0; i < data_size; i += num_threads){
            int index = i + thread_index;
            if(index < data_size){
                out_diff[index] = in_diff[index] * ((in_data[index] > 0)
                                + (in_data[index] <= 0) * 0.01);
            }
        }
    }

    /*
       lrelu backward
       B{ top_[1], top_[0], bottom_[0] } >> *activation_down >> B{ bottom_[1] }; 
     */
    void ActivationDown::compute_gpu(const vector<bool>& add) {
        if(mode_ == "lrelu"){
            Size s = inputs_[0]->size();
            int data_size = s.num() * s.channels() * s.height() * s.width();
            int num_blocks  = std::min(CAFFE_GET_BLOCKS(data_size), 1024);
            int num_threads = CAFFE_CUDA_NUM_THREADS;
            LreluBackward<<< num_blocks, num_threads, 0, stream()>>>(
                inputs_[0]->gpu_data(), 
                inputs_[1]->gpu_data(),
                outputs_[0]->mutable_gpu_data(),
                data_size);
            CUDA_POST_KERNEL_CHECK;
        }
        else if(mode_ == "relu" || mode_ == "sigmoid" || mode_ == "tanh"){
            DTYPE alpha = 1.;
            DTYPE beta = add[0] ? 1. : 0.;
            /*
             */
            CUDNN_CHECK(hipdnnActivationBackward(cudnn_handle(), activation_mode_,
                        &alpha, top_desc_, inputs_[1]->gpu_data(), top_desc_,
                        inputs_[0]->gpu_data(), bottom_desc_, inputs_[2]->gpu_data(),
                        &beta, bottom_desc_, outputs_[0]->mutable_gpu_data()));
        } 
    }

}
