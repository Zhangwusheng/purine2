#include "hip/hip_runtime.h"
#include <math_functions.h>  // CUDA's, not caffe's, for fabs, signbit
#include <thrust/device_vector.h>
#include <thrust/functional.h>  // thrust::plus
#include <thrust/reduce.h>

#include <cmath>
#include <cstdlib>
#include <cstring>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hip/driver_types.h>  // cuda driver types

#include "caffeine/math_functions.hpp"
#include "common/common.hpp"
#include "common/cuda.hpp"

namespace caffe {

using namespace purine;

template <>
void caffe_gpu_gemm<float>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const float alpha, const float* A, const float* B, const float beta,
    float* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasSgemm(cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void caffe_gpu_gemm<double>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const double alpha, const double* A, const double* B, const double beta,
    double* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasDgemm(cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void caffe_gpu_gemv<float>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const float alpha, const float* A, const float* x,
    const float beta, float* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasSgemv(cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

template <>
void caffe_gpu_gemv<double>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const double alpha, const double* A, const double* x,
    const double beta, double* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasDgemv(cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

template <>
void caffe_gpu_axpy<float>(const int N, const float alpha, const float* X,
    float* Y) {
  CUBLAS_CHECK(hipblasSaxpy(cublas_handle(), N, &alpha, X, 1, Y, 1));
}

template <>
void caffe_gpu_axpy<double>(const int N, const double alpha, const double* X,
    double* Y) {
  CUBLAS_CHECK(hipblasDaxpy(cublas_handle(), N, &alpha, X, 1, Y, 1));
}

void caffe_gpu_memcpy(const size_t N, const void* X, void* Y) {
  if (X != Y) {
    CUDA_CHECK(hipMemcpy(Y, X, N, hipMemcpyDefault));  // NOLINT(caffe/alt_fn)
  }
}

template <>
void caffe_gpu_scal<float>(const int N, const float alpha, float *X) {
  CUBLAS_CHECK(hipblasSscal(cublas_handle(), N, &alpha, X, 1));
}

template <>
void caffe_gpu_scal<double>(const int N, const double alpha, double *X) {
  CUBLAS_CHECK(hipblasDscal(cublas_handle(), N, &alpha, X, 1));
}

template <>
void caffe_gpu_axpby<float>(const int N, const float alpha, const float* X,
    const float beta, float* Y) {
  caffe_gpu_scal<float>(N, beta, Y);
  caffe_gpu_axpy<float>(N, alpha, X, Y);
}

template <>
void caffe_gpu_axpby<double>(const int N, const double alpha, const double* X,
    const double beta, double* Y) {
  caffe_gpu_scal<double>(N, beta, Y);
  caffe_gpu_axpy<double>(N, alpha, X, Y);
}

template <>
void caffe_gpu_dot<float>(const int n, const float* x, const float* y,
    float* out) {
  CUBLAS_CHECK(hipblasSdot(cublas_handle(), n, x, 1, y, 1, out));
}

template <>
void caffe_gpu_dot<double>(const int n, const double* x, const double* y,
    double * out) {
  CUBLAS_CHECK(hipblasDdot(cublas_handle(), n, x, 1, y, 1, out));
}

template <>
void caffe_gpu_asum<float>(const int n, const float* x, float* y) {
  CUBLAS_CHECK(hipblasSasum(cublas_handle(), n, x, 1, y));
}

template <>
void caffe_gpu_asum<double>(const int n, const double* x, double* y) {
  CUBLAS_CHECK(hipblasDasum(cublas_handle(), n, x, 1, y));
}

template <>
void caffe_gpu_scale<float>(const int n, const float alpha, const float *x,
                            float* y) {
  CUBLAS_CHECK(hipblasScopy(cublas_handle(), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasSscal(cublas_handle(), n, &alpha, y, 1));
}

template <>
void caffe_gpu_scale<double>(const int n, const double alpha, const double *x,
                             double* y) {
  CUBLAS_CHECK(hipblasDcopy(cublas_handle(), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasDscal(cublas_handle(), n, &alpha, y, 1));
}

__global__ void scale(float* data, const float* alpha, const int n){
    float a = alpha[0];
    CUDA_KERNEL_LOOP(index, n) {
        data[index] /= a;
    }
}

void gpu_scale(float *data, const float* alpha, const int len){
    const int N = len;
    scale<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, stream()>>>(data,
            alpha, N);
}

template <typename Dtype>
__global__ void set_kernel(const int n, const Dtype alpha, Dtype* y) {
    CUDA_KERNEL_LOOP(index, n) {
        y[index] = alpha;
    }
}

template <typename Dtype>
void caffe_gpu_set(const int N, const Dtype alpha, Dtype* Y) {
    if (alpha == 0) {
        CUDA_CHECK(hipMemsetAsync(Y, 0, sizeof(Dtype) * N, stream()));
        return;
    }
    // NOLINT_NEXT_LINE(whitespace/operators)
    set_kernel<Dtype>
        <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, stream()>>>(
                N, alpha, Y);
}

template void caffe_gpu_set<int>(const int N, const int alpha, int* Y);
template void caffe_gpu_set<float>(const int N, const float alpha, float* Y);
template void caffe_gpu_set<double>(const int N, const double alpha, double* Y);

template <typename Dtype>
__global__ void add_scalar_kernel(const int n, const Dtype alpha, Dtype* y) {
    CUDA_KERNEL_LOOP(index, n) {
        y[index] += alpha;
    }
}

template <>
void caffe_gpu_add_scalar(const int N, const float alpha, float* Y) {
    // NOLINT_NEXT_LINE(whitespace/operators)
    add_scalar_kernel<float>
        <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, stream()>>>(
                N, alpha, Y);
}

template <>
void caffe_gpu_add_scalar(const int N, const double alpha, double* Y) {
    // NOLINT_NEXT_LINE(whitespace/operators)
    add_scalar_kernel<double>
        <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, stream()>>>(
                N, alpha, Y);
}

template <typename Dtype>
__global__ void add_kernel(const int n, const Dtype* a,
        const Dtype* b, Dtype* y) {
    CUDA_KERNEL_LOOP(index, n) {
        y[index] = a[index] + b[index];
    }
}

template <>
void caffe_gpu_add<float>(const int N, const float* a, const float* b,
        float* y) {
    // NOLINT_NEXT_LINE(whitespace/operators)
    add_kernel<float>
        <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, stream()>>>(
                N, a, b, y);
}

template <>
void caffe_gpu_add<double>(const int N, const double* a, const double* b,
        double* y) {
    // NOLINT_NEXT_LINE(whitespace/operators)
    add_kernel<double>
        <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, stream()>>>(
                N, a, b, y);
}

template <typename Dtype>
__global__ void sub_kernel(const int n, const Dtype* a,
        const Dtype* b, Dtype* y) {
    CUDA_KERNEL_LOOP(index, n) {
        y[index] = a[index] - b[index];
    }
}

template <>
void caffe_gpu_sub<float>(const int N, const float* a, const float* b,
        float* y) {
    // NOLINT_NEXT_LINE(whitespace/operators)
    sub_kernel<float>
        <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, stream()>>>(
                N, a, b, y);
}

template <>
void caffe_gpu_sub<double>(const int N, const double* a, const double* b,
        double* y) {
    // NOLINT_NEXT_LINE(whitespace/operators)
    sub_kernel<double>
        <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, stream()>>>(
                N, a, b, y);
}

template <typename Dtype>
__global__ void mul_kernel(const int n, const Dtype* a,
        const Dtype* b, Dtype* y) {
    CUDA_KERNEL_LOOP(index, n) {
        y[index] = a[index] * b[index];
    }
}

template <>
void caffe_gpu_mul<float>(const int N, const float* a,
        const float* b, float* y) {
    // NOLINT_NEXT_LINE(whitespace/operators)
    mul_kernel<float>
        <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, stream()>>>(
                N, a, b, y);
}

template <>
void caffe_gpu_mul<double>(const int N, const double* a,
        const double* b, double* y) {
    // NOLINT_NEXT_LINE(whitespace/operators)
    mul_kernel<double>
        <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, stream()>>>(
                N, a, b, y);
}

template <typename Dtype>
__global__ void div_kernel(const int n, const Dtype* a,
        const Dtype* b, Dtype* y) {
    CUDA_KERNEL_LOOP(index, n) {
        y[index] = a[index] / b[index];
    }
}

template <>
void caffe_gpu_div<float>(const int N, const float* a,
        const float* b, float* y) {
    // NOLINT_NEXT_LINE(whitespace/operators)
    div_kernel<float>
        <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, stream()>>>(
                N, a, b, y);
}

template <>
void caffe_gpu_div<double>(const int N, const double* a,
        const double* b, double* y) {
    // NOLINT_NEXT_LINE(whitespace/operators)
    div_kernel<double>
        <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, stream()>>>(
                N, a, b, y);
}

template <typename Dtype>
__global__ void abs_kernel(const int n, const Dtype* a, Dtype* y) {
    CUDA_KERNEL_LOOP(index, n) {
        y[index] = abs(a[index]);
    }
}

template <>
void caffe_gpu_abs<float>(const int N, const float* a, float* y) {
    // NOLINT_NEXT_LINE(whitespace/operators)
    abs_kernel<float>
        <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, stream()>>>(
                N, a, y);
}

template <>
void caffe_gpu_abs<double>(const int N, const double* a, double* y) {
    // NOLINT_NEXT_LINE(whitespace/operators)
    abs_kernel<double>
        <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, stream()>>>(
                N, a, y);
}


template <typename Dtype>
__global__ void powx_kernel(const int n, const Dtype* a,
        const Dtype alpha, Dtype* y) {
    CUDA_KERNEL_LOOP(index, n) {
        y[index] = pow(a[index], alpha);
    }
}

template <>
void caffe_gpu_powx<float>(const int N, const float* a,
        const float alpha, float* y) {
    // NOLINT_NEXT_LINE(whitespace/operators)
    powx_kernel<float>
        <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, stream()>>>(
                N, a, alpha, y);
}

template <>
void caffe_gpu_powx<double>(const int N, const double* a,
        const double alpha, double* y) {
    // NOLINT_NEXT_LINE(whitespace/operators)
    powx_kernel<double>
        <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, stream()>>>(
                N, a, alpha, y);
}

DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sign, y[index] = (Dtype(0) < x[index])
        - (x[index] < Dtype(0)));
    DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sgnbit, y[index] = signbit(x[index]));

    __global__ void popc_kernel(const int n, const float* a,
            const float* b, uint8_t* y) {
        CUDA_KERNEL_LOOP(index, n) {
            y[index] = __popc(static_cast<uint32_t>(a[index]) ^
                    static_cast<uint32_t>(b[index]));
        }
    }

__global__ void popcll_kernel(const int n, const double* a,
        const double* b, uint8_t* y) {
    CUDA_KERNEL_LOOP(index, n) {
        y[index] = __popcll(static_cast<uint64_t>(a[index]) ^
                static_cast<uint64_t>(b[index]));
    }
}

template <>
uint32_t caffe_gpu_hamming_distance<float>(const int n, const float* x,
        const float* y) {
    // TODO: Fix caffe_gpu_hamming_distance (see failing unit test
    // TestHammingDistanceGPU in test_math_functions.cpp).
    NOT_IMPLEMENTED;
    thrust::device_vector<uint8_t> popcounts(n);
    // NOLINT_NEXT_LINE(whitespace/operators)
    popc_kernel<<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS, 0, stream()>>>(
            n, x, y, thrust::raw_pointer_cast(popcounts.data()));
    return thrust::reduce(popcounts.begin(), popcounts.end(),
            (uint32_t) 0, thrust::plus<uint32_t>());
}

template <>
uint32_t caffe_gpu_hamming_distance<double>(const int n, const double* x,
        const double* y) {
    // TODO: Fix caffe_gpu_hamming_distance (see failing unit test
    // TestHammingDistanceGPU in test_math_functions.cpp).
    NOT_IMPLEMENTED;
    thrust::device_vector<uint8_t> popcounts(n);
    // NOLINT_NEXT_LINE(whitespace/operators)
    popcll_kernel<<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS, 0, stream()>>>(
            n, x, y, thrust::raw_pointer_cast(popcounts.data()));
    return thrust::reduce(popcounts.begin(), popcounts.end(),
            /* NOLINT_NEXT_LINE(build/include_what_you_use) */
            (uint32_t) 0, thrust::plus<uint32_t>());
}

void caffe_gpu_rng_uniform(const int n, unsigned int* r) {
    CURAND_CHECK(hiprandGenerate(curand_generator(), r, n));
}

template <>
void caffe_gpu_rng_uniform<float>(const int n, const float a, const float b,
        float* r) {
    CURAND_CHECK(hiprandGenerateUniform(curand_generator(), r, n));
    const float range = b - a;
    if (range != static_cast<float>(1)) {
        caffe_gpu_scal(n, range, r);
    }
    if (a != static_cast<float>(0)) {
        caffe_gpu_add_scalar(n, a, r);
    }
}

template <>
void caffe_gpu_rng_uniform<double>(const int n, const double a, const double b,
        double* r) {
    CURAND_CHECK(hiprandGenerateUniformDouble(curand_generator(), r, n));
    const double range = b - a;
    if (range != static_cast<double>(1)) {
        caffe_gpu_scal(n, range, r);
    }
    if (a != static_cast<double>(0)) {
        caffe_gpu_add_scalar(n, a, r);
    }
}

template <>
void caffe_gpu_rng_gaussian(const int n, const float mu, const float sigma,
        float* r) {
    CURAND_CHECK(
            hiprandGenerateNormal(curand_generator(), r, n, mu, sigma));
}

template <>
void caffe_gpu_rng_gaussian(const int n, const double mu, const double sigma,
        double* r) {
    CURAND_CHECK(
            hiprandGenerateNormalDouble(curand_generator(), r, n, mu, sigma));
}

template <typename Dtype>
__global__ void threshold_kernel(const int n, Dtype* a, Dtype threshold) {
    CUDA_KERNEL_LOOP(index, n) {
        a[index] = a[index] > threshold ? ((Dtype) 1.0) : ((Dtype) 0.0);
    }
}

template <>
void caffe_gpu_rng_bernoulli(const int n, const float p, float* r) {
    CURAND_CHECK(hiprandGenerateUniform(curand_generator(), r, n));
    CHECK_GE(n, 0);
    CHECK(r);
    CHECK_GE(p, 0);
    CHECK_LE(p, 1);
    threshold_kernel<float>
        <<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS, 0, stream()>>> (
                n, r, 1 - p);
}

template <>
void caffe_gpu_rng_bernoulli(const int n, const double p, double* r) {
    CURAND_CHECK(hiprandGenerateUniformDouble(curand_generator(), r, n));
    CHECK_GE(n, 0);
    CHECK(r);
    CHECK_GE(p, 0);
    CHECK_LE(p, 1);
    threshold_kernel<double>
        <<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS, 0, stream()>>> (
                n, r, 1 - p);
}

template <typename Dtype>
void caffe_gpu_copy(int N, const Dtype* X, Dtype* Y) {
    if (X != Y) {
        CUDA_CHECK(hipMemcpy(Y, X, sizeof(Dtype) * N, hipMemcpyDefault));
    }
}

template void caffe_gpu_copy<int>(const int N, const int* X, int* Y);
template void caffe_gpu_copy<unsigned int>(const int N, const unsigned int* X,
        unsigned int* Y);
template void caffe_gpu_copy<float>(const int N, const float* X, float* Y);
template void caffe_gpu_copy<double>(const int N, const double* X, double* Y);

bool purine_gpu_compare(const DTYPE* array1, const DTYPE* array2, int count) {
    DTYPE* a;
    CUDA_CHECK(hipHostMalloc(&a, sizeof(DTYPE) * count, hipHostMallocPortable));
    DTYPE* b;
    CUDA_CHECK(hipHostMalloc(&b, sizeof(DTYPE) * count, hipHostMallocPortable));
    CUDA_CHECK(hipMemcpy(a, array1, sizeof(DTYPE) * count, hipMemcpyDefault));
    CUDA_CHECK(hipMemcpy(b, array2, sizeof(DTYPE) * count, hipMemcpyDefault));
    bool ret = purine_cpu_compare(a, b, count);
    CUDA_CHECK(hipHostFree(a));
    CUDA_CHECK(hipHostFree(b));
    return ret;
}

}  // namespace caffe
